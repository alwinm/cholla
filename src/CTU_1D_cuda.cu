#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"



Real CTU_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt, int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if ( !memory_allocated ) {

    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( hipHostMalloc(&host_dti_array, ngrid*sizeof(Real)));
    #ifdef COOLING_GPU
    CudaSafeCall( hipHostMalloc(&host_dt_array, ngrid*sizeof(Real)));
    #endif

    // allocate memory on the GPU
    CudaSafeCall( hipMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( hipMalloc((void**)&Q_Lx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( hipMalloc((void**)&Q_Rx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( hipMalloc((void**)&F_x,   (n_fields)*n_cells*sizeof(Real)) );
    CudaSafeCall( hipMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #if defined COOLING_GPU
    CudaSafeCall( hipMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif  

    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }

  // copy the conserved variable array onto the GPU
  CudaSafeCall( hipMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), hipMemcpyHostToDevice) );
  CudaCheckError();


  // Step 1: Do the reconstruction
  #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMC
  hipLaunchKernelGGL(PPMC_cuda, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif

  
  // Step 2: Calculate the fluxes
  #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim3(dimGrid), dim3(dimBlock), 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim3(dimGrid), dim3(dimBlock), 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim3(dimGrid), dim3(dimBlock), 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();

  #ifdef DE
  // Compute the divergence of Vel before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables
  Partial_Update_Advected_Internal_Energy_1D<<<dimGrid,dimBlock>>>( dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, dx, dt, gama, n_fields );
  #endif


  // Step 3: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_1D, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, F_x, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();
   

  // Sychronize the total and internal energy, if using dual-energy formalism
  #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_1D, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, n_cells, n_ghost, n_fields);
  hipLaunchKernelGGL(Sync_Energies_1D, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, n_cells, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif


  // Apply cooling
  #ifdef COOLING_GPU
  hipLaunchKernelGGL(cooling_kernel, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gama, dev_dti_array);
  CudaCheckError();
  #endif

  // Calculate the next timestep
  hipLaunchKernelGGL(Calc_dt_1D, dim3(dimGrid), dim3(dimBlock), 0, 0, dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( hipMemcpy(host_conserved1, dev_conserved, n_fields*n_cells*sizeof(Real), hipMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( hipMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), hipMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #if defined COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( hipMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), hipMemcpyDeviceToHost) );
  // find maximum inverse timestep from cooling time
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  if (min_dt < C_cfl/max_dti) {
    max_dti = C_cfl/min_dt;
  }
  #endif

  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_CTU_1D();
  #endif
  

  // return the maximum inverse timestep
  return max_dti;


}

void Free_Memory_CTU_1D() {

  // free the CPU memory
  CudaSafeCall( hipHostFree(host_dti_array));
  #if defined COOLING_GPU
  CudaSafeCall( hipHostFree(host_dt_array));
  #endif

  // free the GPU memory
  hipFree(dev_conserved);
  hipFree(Q_Lx);
  hipFree(Q_Rx);
  hipFree(F_x);
  hipFree(dev_dti_array);
  #if defined COOLING_GPU
  hipFree(dev_dt_array);
  #endif

}


#endif //CUDA
