#include "hip/hip_runtime.h"
/*!
 * \file reduction_utilities_tests.cpp
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Tests for the contents of reduction_utilities.h and reduction_utilities.cpp
 *
 */

// STL Includes
#include <vector>
#include <string>
#include <iostream>
#include <random>

// External Includes
#include <gtest/gtest.h>    // Include GoogleTest and related libraries/headers

// Local Includes
#include "../utils/testing_utilities.h"
#include "../utils/reduction_utilities.h"
#include "../global/global.h"

// =============================================================================
// Tests for divergence max reduction
// =============================================================================
TEST(tALLKernelReduceMax, CorrectInputExpectCorrectOutput)
{
    // Launch parameters
    // =================
    uint numBlocks, threadsPerBlock;
    reduction_utilities::reductionLaunchParams(numBlocks, threadsPerBlock);

    // Grid Parameters & testing parameters
    // ====================================
    size_t const gridSize = 128;
    size_t const size     = std::pow(gridSize, 3);;
    Real   const maxValue = 4;
    std::vector<Real> host_grid(size);
    Real host_max;

    // Fill grid with random values and assign maximum value
    std::mt19937 prng(1);
    std::uniform_real_distribution<double> doubleRand(-std::abs(maxValue)-1, std::abs(maxValue) - 1);
    std::uniform_int_distribution<int> intRand(0, host_grid.size()-1);
    for (size_t i = 0; i < host_grid.size(); i++)
    {
        host_grid.at(i) = doubleRand(prng);
    }
    host_grid.at(intRand(prng)) = maxValue;


    // Allocating and copying to device
    // ================================
    Real *dev_grid, *dev_max;
    CudaSafeCall(hipMalloc(&dev_grid, host_grid.size() * sizeof(Real)));
    CudaSafeCall(hipMalloc(&dev_max, sizeof(Real)));
    CudaSafeCall(hipMemcpy(dev_grid, host_grid.data(), host_grid.size() * sizeof(Real), hipMemcpyHostToDevice));

    // Do the reduction
    // ================
    hipLaunchKernelGGL(reduction_utilities::kernelReduceMax, numBlocks, threadsPerBlock, 0, 0, dev_grid, dev_max, host_grid.size());
    CudaCheckError();

    // Copy back and sync
    // ==================
    CudaSafeCall(hipMemcpy(&host_max, dev_max, sizeof(Real), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // Perform comparison
    testingUtilities::checkResults(maxValue, host_max, "maximum value found");
}
// =============================================================================
// Tests for divergence max reduction
// =============================================================================
