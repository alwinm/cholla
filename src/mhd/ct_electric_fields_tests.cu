/*!
 * \file ct_electric_fields_tests.cu
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief Tests for the CT electric fields
 *
 */

// STL Includes
#include <vector>
#include <string>
#include <iostream>
#include <numeric>
#include <cmath>

// External Includes
#include <gtest/gtest.h>    // Include GoogleTest and related libraries/headers

// Local Includes
#include "../utils/testing_utilities.h"
#include "../mhd/ct_electric_fields.h"
#include "../global/global.h"

#ifdef MHD
// =============================================================================
// Tests for the mhd::Calculate_CT_Electric_Fields kernel
// =============================================================================

// =============================================================================
/*!
 * \brief Test fixture for tMHDCalculateCTElectricFields test suite
 *
 */
class tMHDCalculateCTElectricFields : public ::testing::Test
{
public:

    /*!
     * \brief Initialize and allocate all the various required variables and
    * arrays
     *
     */
    tMHDCalculateCTElectricFields()
        :
        nx(3),
        ny(nx),
        nz(nx),
        n_cells(nx*ny*nz),
        fluxX(n_cells * (grid_enum::num_flux_fields)),
        fluxY(n_cells * (grid_enum::num_flux_fields)),
        fluxZ(n_cells * (grid_enum::num_flux_fields)),
        grid (n_cells * (grid_enum::num_fields)),
        testCTElectricFields(n_cells * 3, -999.),
        fiducialData(n_cells * 3, -999.),
        dimGrid((n_cells + TPB - 1),1,1),
        dimBlock(TPB,1,1)
    {
        // Allocate device arrays
        CudaSafeCall ( hipMalloc(&dev_fluxX, fluxX.size()*sizeof(double)) );
        CudaSafeCall ( hipMalloc(&dev_fluxY, fluxY.size()*sizeof(double)) );
        CudaSafeCall ( hipMalloc(&dev_fluxZ, fluxZ.size()*sizeof(double)) );
        CudaSafeCall ( hipMalloc(&dev_grid,   grid.size()*sizeof(double)) );
        CudaSafeCall ( hipMalloc(&dev_testCTElectricFields, testCTElectricFields.size()*sizeof(double)) );

        // Populate the grids with values where vector.at(i) = double(i). The
        // values chosen aren't that important, just that every cell has a unique
        // value
        std::iota(std::begin(fluxX), std::end(fluxX), 0.);
        std::iota(std::begin(fluxY), std::end(fluxY), fluxX.back() + 1);
        std::iota(std::begin(fluxZ), std::end(fluxZ), fluxY.back() + 1);
        std::iota(std::begin(grid),  std::end(grid),  fluxZ.back() + 1);
    }
    ~tMHDCalculateCTElectricFields() = default;
protected:
    // Initialize the test grid and other state variables
    size_t const nx, ny, nz;
    size_t const n_cells;

    // Launch Parameters
    dim3 const dimGrid;  // How many blocks in the grid
    dim3 const dimBlock;  // How many threads per block

    // Make sure the vector is large enough that the locations where the
    // magnetic field would be in the real grid are filled
    std::vector<double> fluxX;
    std::vector<double> fluxY;
    std::vector<double> fluxZ;
    std::vector<double> grid;
    std::vector<double> testCTElectricFields;
    std::vector<double> fiducialData;

    // device pointers
    double *dev_fluxX, *dev_fluxY, *dev_fluxZ, *dev_grid, *dev_testCTElectricFields;

    /*!
     * \brief Launch the kernel and check results
     *
     */
    void runTest()
    {
        // Copy values to GPU
        CudaSafeCall( hipMemcpy(dev_fluxX, fluxX.data(), fluxX.size()*sizeof(Real), hipMemcpyHostToDevice) );
        CudaSafeCall( hipMemcpy(dev_fluxY, fluxY.data(), fluxY.size()*sizeof(Real), hipMemcpyHostToDevice) );
        CudaSafeCall( hipMemcpy(dev_fluxZ, fluxZ.data(), fluxZ.size()*sizeof(Real), hipMemcpyHostToDevice) );
        CudaSafeCall( hipMemcpy(dev_grid,  grid.data(),   grid.size()*sizeof(Real), hipMemcpyHostToDevice) );
        CudaSafeCall( hipMemcpy(dev_testCTElectricFields,
                                 testCTElectricFields.data(),
                                 testCTElectricFields.size()*sizeof(Real),
                                 hipMemcpyHostToDevice) );

        // Call the kernel to test
        hipLaunchKernelGGL(mhd::Calculate_CT_Electric_Fields,
                           dimGrid,
                           dimBlock,
                           0,
                           0,
                           dev_fluxX,
                           dev_fluxY,
                           dev_fluxZ,
                           dev_grid,
                           dev_testCTElectricFields,
                           nx,
                           ny,
                           nz,
                           n_cells);
        CudaCheckError();

        // Copy test data back
        CudaSafeCall( hipMemcpy(testCTElectricFields.data(),
                                 dev_testCTElectricFields,
                                 testCTElectricFields.size()*sizeof(Real),
                                 hipMemcpyDeviceToHost) );
        hipDeviceSynchronize();

        // Check the results
        for (size_t i = 0; i < fiducialData.size(); i++)
        {
            int xid, yid, zid;
            cuda_utilities::compute3DIndices(i, nx, ny, xid, yid, zid);
            testingUtilities::checkResults(fiducialData.at(i),
                                           testCTElectricFields.at(i),
                                           "value at i = " + std::to_string(i)
                                           + ", xid  = " + std::to_string(xid)
                                           + ", yid  = " + std::to_string(yid)
                                           + ", zid  = " + std::to_string(zid));
        }
    }
};
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields,
       PositiveVelocityExpectCorrectOutput)
{
    // Fiducial values
    fiducialData.at(26) =  206.29859653255295;
    fiducialData.at(53) = -334.90052254763339;
    fiducialData.at(80) =  209.53472440298236;

    // Launch kernel and check results
    runTest();
}
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields,
       NegativeVelocityExpectCorrectOutput)
{
    // Fiducial values
    fiducialData.at(26) =  203.35149422304994;
    fiducialData.at(53) = -330.9860399765279;
    fiducialData.at(80) =  208.55149905461991;

    // Set the density fluxes to be negative to indicate a negative velocity
    // across the face
    for (size_t i = 0; i < n_cells; i++)
    {
        fluxX.at(i) = -fluxX.at(i);
        fluxY.at(i) = -fluxY.at(i);
        fluxZ.at(i) = -fluxZ.at(i);
    }

    // Launch kernel and check results
    runTest();
}
// =============================================================================

// =============================================================================
TEST_F(tMHDCalculateCTElectricFields,
       ZeroVelocityExpectCorrectOutput)
{
    // Fiducial values
    fiducialData.at(26) =  204.82504537780144;
    fiducialData.at(53) = -332.94328126208063;
    fiducialData.at(80) =  209.04311172880114;

    // Set the density fluxes to be negative to indicate a negative velocity
    // across the face
    for (size_t i = 0; i < n_cells; i++)
    {
        fluxX.at(i) = 0.0;
        fluxY.at(i) = 0.0;
        fluxZ.at(i) = 0.0;
    }

    // Launch kernel and check results
    runTest();
}
// =============================================================================
#endif // MHD
